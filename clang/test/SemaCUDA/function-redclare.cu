// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only \
// RUN:   -isystem %S/Inputs -verify %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fsyntax-only \
// RUN:   -isystem %S/Inputs -fcuda-is-device -verify %s
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only \
// RUN:   -isystem %S/Inputs -verify=redecl -Woffload-incompatible-redeclare %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fsyntax-only \
// RUN:   -isystem %S/Inputs -fcuda-is-device -Woffload-incompatible-redeclare -verify=redecl %s

// expected-no-diagnostics
#include "hip/hip_runtime.h"

__device__ void f(); // redecl-note {{previous declaration is here}}

void f() {} // redecl-warning {{incompatible host/device attribute with redeclaration: new declaration is __host__ function, old declaration is __device__ function. It will cause warning with nvcc}}

void g(); // redecl-note {{previous declaration is here}}

__device__ void g() {} // redecl-warning {{incompatible host/device attribute with redeclaration: new declaration is __device__ function, old declaration is __host__ function. It will cause warning with nvcc}}
