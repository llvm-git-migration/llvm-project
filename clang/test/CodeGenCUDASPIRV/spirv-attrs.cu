#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fcuda-is-device -triple spirv64 -o - -emit-llvm -x cuda %s  | FileCheck %s
// RUN: %clang_cc1 -fcuda-is-device -triple spirv32 -o - -emit-llvm -x cuda %s  | FileCheck %s

#define __global__ __attribute__((global))

__attribute__((reqd_work_group_size(128, 1, 1)))
__global__ void reqd_work_group_size_128_1_1() {}
// CHECK: define spir_kernel void @_Z28reqd_work_group_size_128_1_1v() #[[ATTR:[0-9]+]] !reqd_work_group_size ![[SIZE_128:.*]]

__attribute__((work_group_size_hint(2, 2, 2)))
__global__ void work_group_size_hint_2_2_2() {}
// CHECK: define spir_kernel void @_Z26work_group_size_hint_2_2_2v() #[[ATTR]] !work_group_size_hint ![[HINT_2:.*]]

__attribute__((vec_type_hint(int)))
__global__ void vec_type_hint_int() {}
// CHECK: define spir_kernel void @_Z17vec_type_hint_intv() #[[ATTR]] !vec_type_hint ![[VEC_HINT:.*]]

__attribute__((intel_reqd_sub_group_size(64)))
__global__ void intel_reqd_sub_group_size_64() {}
// CHECK: define spir_kernel void @_Z28intel_reqd_sub_group_size_64v() #[[ATTR]] !intel_reqd_sub_group_size ![[SUB_GROUP:.*]]

// CHECK: attributes #[[ATTR]] = { convergent mustprogress noinline norecurse nounwind optnone {{.*}} }

// CHECK: ![[SIZE_128]] = !{i32 128, i32 1, i32 1}
// CHECK: ![[HINT_2]] = !{i32 2, i32 2, i32 2}
// CHECK: ![[VEC_HINT]] = !{i32 undef, i32 1}
// CHECK: ![[SUB_GROUP]] = !{i32 64}

